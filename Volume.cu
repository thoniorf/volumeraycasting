#include "hip/hip_runtime.h"
/* Volume
 * Volumetric data
*/
#ifndef VOLUME
#define VOLUME
#endif // !VOLUME
#define _USE_MATH_DEFINES
//#define M_PI 3.14159265359
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <string>
#include <vector>
#include <ctime>
#include <math.h>
#include <iostream>
#include <stdio.h>
#include <fstream>
#include <exception>

#include "hip/hip_runtime.h"
#include ""

std::time_t timer;
int timeout = 300;

struct Options {
public:
	double threshold = 0;
	double minIntensity = 0;
	double maxIntensity = 0;
	double scale = 1;
	double viewOffset = 1;
	size_t imageWidth = 2;
	size_t imageHeight = 2;
	size_t imageDimension = 4;
	double fov = 60;
	double imageAspectRatio = 1;
	double visibleObjectsSize = 0;

	void setImageSize(double width, double height) {
		imageWidth = std::floor(width);
		imageHeight = std::floor(height);
	}

	void setIntensity(double min, double max) {
		minIntensity = min;
		maxIntensity = max;
	}

};
__host__ __device__
inline
double degToRad(const double& d) {
	return d * M_PI / 180;
}
__host__ __device__
inline
double fixZeroDoublePrecisionError(const double& x) {
	if (std::fabs(x) <= 5.00e-5)
		return 0;
	return x;
}
/* 3D VECTOR CLASS */
class Vector3 {
public:
	double x, y, z;
	__host__ __device__
	Vector3() { x = y = z = 0; }
	__host__ __device__
	Vector3(double scalar) : x(scalar), y(scalar), z(scalar) {}
	__host__ __device__
	Vector3(double i, double j, double k) : x(fixZeroDoublePrecisionError(i)), y(fixZeroDoublePrecisionError(j)), z(fixZeroDoublePrecisionError(k)) {}
	__host__ __device__
	const double& operator [] (size_t i) const { return (&x)[i]; }
	__host__ __device__
	double& operator [] (size_t i) { return (&x)[i]; }
	__host__ __device__
	Vector3 operator - () { return Vector3(-x, -y, -z); }
	__host__ __device__
	Vector3 operator + (Vector3 v) { return Vector3(x + v.x, y + v.y, z + v.z); }
	__host__ __device__
	Vector3 operator - (Vector3 v) { return Vector3(x - v.x, y - v.y, z - v.z); }
	__host__ __device__
	Vector3 operator * (Vector3 v) { return Vector3(x * v.x, y * v.y, z * v.z); }
	__host__ __device__
	Vector3 operator * (double scalar) { return Vector3(x * scalar, y * scalar, z * scalar); }
	__host__ __device__
	Vector3 operator / (double scalar) { return Vector3(x / scalar, y / scalar, z / scalar); }
	__host__ __device__
	Vector3 operator = (Vector3 v) {
		x = fixZeroDoublePrecisionError(v.x);
		y = fixZeroDoublePrecisionError(v.y);
		z = fixZeroDoublePrecisionError(v.z);
		return Vector3(x, y, z);
	}
	__host__ __device__
	bool operator != (Vector3 v) {
		return (v.x != x || v.y != y || v.z != z);
	}
	__host__ __device__
	inline
		double norm() const { return x * x + y * y + z * z; }
	__host__ __device__	
	inline
		double length() const { return sqrt(norm()); }
	__host__ __device__
	inline
		Vector3 normalize() {
		double n = norm();
		double xx = x, yy = y, zz = z;
		if (n > 0) {
			double factor = 1 / sqrt(n);
			xx *= factor, yy *= factor, zz *= factor;
		}
		return Vector3(xx, yy, zz);
	}
	__host__ __device__
	inline
		double dot(const Vector3& v) {
		return x * v.x + y * v.y + z * v.z;
	}
	__host__ __device__
	inline
		Vector3 cross(const Vector3& v) {
		double cx, cy, cz;
		cx = y * v.z - z * v.y;
		cy = z * v.x - x * v.z;
		cz = x * v.y - y * v.x;
		return Vector3(cx, cy, cz);
	}
	__host__ __device__
	friend Vector3 operator / (double scalar, Vector3 vec) {
		return Vector3(scalar / vec.x, scalar / vec.y, scalar / vec.z);
	}
	__host__ 
	friend std::ostream& operator << (std::ostream &s, const Vector3 &v)
	{
		return s << "(" << v.x << " , " << v.y << " , " << v.z << ")";
	}

};

class Matrix4x4 {
	/*
	x1 x2 x3 0
	y1 y2 y3 0
	z1 z2 z3 0
	tx ty tz 1

	L = T * R * S
	scale , rotate, translate
	*/
private:
	double x[4][4] = { {1,0,0,0},{0,1,0,0},{0,0,1,0},{0,0,0,1} };
public:
	__host__ __device__
	Matrix4x4() {}
	__host__ __device__
	Matrix4x4(Vector3 translation) {
		// translation component
		x[0][3] = translation.x;
		x[1][3] = translation.y;
		x[2][3] = translation.z;

	}
	__host__ __device__
	Matrix4x4(double c00, double c01, double c02, double c03, double c10, double c11, double c12, double c13, double c20, double c21, double c22, double c23, double c30, double c31, double c32, double c33) {
		x[0][0] = c00;
		x[0][1] = c01;
		x[0][2] = c02;
		x[0][3] = c03;
		x[1][0] = c10;
		x[1][1] = c11;
		x[1][2] = c12;
		x[1][3] = c13;
		x[2][0] = c20;
		x[2][1] = c21;
		x[2][2] = c22;
		x[2][3] = c23;
		x[3][0] = c30;
		x[3][1] = c31;
		x[3][2] = c32;
		x[3][3] = c33;
	}
	__host__ __device__
	const double* operator [] (uint8_t i) const { return x[i]; }
	__host__ __device__
	double* operator [] (uint8_t i) { return x[i]; }
	__host__ __device__
	inline
		Matrix4x4 matrixMulti(const Matrix4x4& m) {
		Matrix4x4 c;
		for (uint8_t i = 0; i < 4; ++i) {
			for (uint8_t j = 0; j < 4; ++j) {
				c[i][j] = x[i][0] * m[0][j] + x[i][1] * m[1][j] + x[i][2] * m[2][j] + x[i][3] * m[3][j];
			}
		}
		return c;
	}
	__host__ __device__
	inline
		Vector3 VecMatrixMulti(const Vector3& src) {
		double a, b, c, w;

		a = src.x * x[0][0] + src.y * x[1][0] + src.z * x[2][0] + x[3][0];
		b = src.x * x[0][1] + src.y * x[1][1] + src.z * x[2][1] + x[3][1];
		c = src.x * x[0][2] + src.y * x[1][2] + src.z * x[2][2] + x[3][2];
		w = src.x * x[0][3] + src.y * x[1][3] + src.z * x[2][3] + x[3][3];
		return Vector3(a / w, b / w, c / w);
	}
	__host__ __device__
	inline
		Vector3 DirMatrixMulti(const Vector3& src) const
	{
		double a, b, c;

		a = src.x * x[0][0] + src.y * x[1][0] + src.z * x[2][0];
		b = src.x * x[0][1] + src.y * x[1][1] + src.z * x[2][1];
		c = src.x * x[0][2] + src.y * x[1][2] + src.z * x[2][2];

		return Vector3(a, b, c);
	}
	__host__ __device__
	inline
		Vector3 ColMajMatrixMulti(const Vector3& src) const
	{
		double a, b, c;

		a = src.x * x[0][0] + src.y * x[0][1] + src.z * x[0][2];
		b = src.x * x[1][0] + src.y * x[1][1] + src.z * x[1][2];
		c = src.x * x[2][0] + src.y * x[2][1] + src.z * x[2][2];

		return Vector3(a, b, c);
	}
	__host__
	friend std::ostream& operator << (std::ostream &s, const Matrix4x4 &m)
	{

		s << "[" << m[0][0] <<
			" " << m[0][1] <<
			" " << m[0][2] <<
			" " << m[0][3] << "\n" <<

			" " << m[1][0] <<
			" " << m[1][1] <<
			" " << m[1][2] <<
			" " << m[1][3] << "\n" <<

			" " << m[2][0] <<
			" " << m[2][1] <<
			" " << m[2][2] <<
			" " << m[2][3] << "\n" <<

			" " << m[3][0] <<
			" " << m[3][1] <<
			" " << m[3][2] <<
			" " << m[3][3] << "]";

		return s;
	}
};

class Color {
protected:
	Vector3 arancione;
	Vector3 rosso;
	Vector3 giallo;
	Vector3 fucsia;
	Vector3 verdino;
	Vector3 rosa;
	Vector3 azzurro;
	Vector3 default;
public:
	__host__
	Color() {
		arancione = Vector3(1, 0.65, 0);
		rosso = Vector3(1, 0, 0);
		giallo = Vector3(1, 1, 0);
		fucsia = Vector3(0.65, 0.12, 0.94);
		verdino = Vector3(0.48, 0.99, 0);
		rosa = Vector3(1, 0.75, 0.80);
		azzurro = Vector3(0.68, 0.85, 0.90);
		default = Vector3(0.5, 0.5, 0.5);
	}
	__device__
	Vector3 getColorByIndex(size_t index) {
		switch (index)
		{
		case 0: return arancione;
		case 1: return rosso;
		case 2: return giallo;
		case 3: return fucsia;
		case 4: return verdino;
		case 5: return rosa;
		case 6: return azzurro;
		default:
			return default;
		}
	}
};
class YuvColor : public Color {
public:
	__host__
	YuvColor() {
		arancione = Vector3(1, -0.318368, 0.251903);
		rosso = Vector3(1, -0.09991, 0.615);
		giallo = Vector3(1, -0.436, 0.05639);
		fucsia = Vector3(1, 0.304568, 0.27971);
		verdino = Vector3(1, -0.380686, -0.257824);
		rosa = Vector3(1, -0.0031775, 0.15093);
		azzurro = Vector3(1, 0.0387847, -0.10737);
		default = Vector3(1, 0, 0);
	}
};

/* AABBOX CLASS */
class Grid {
public:
	Vector3 bounds[2] = { 0 };
	__host__ __device__
	Grid() {}
	__host__ __device__
	Grid(const Vector3 &min, const Vector3 &max) {
		bounds[0] = min, bounds[1] = max;
	}
	__device__
	bool isOutsideGrid(int ix, int iy, int iz) {
		return ix < 0 || iy < 0 || iz < 0 || ix > bounds[1].x - 1 || iy > bounds[1].y - 1 || iz > bounds[1].z - 1;
	}
	__device__
	inline
		bool isInsideGrid(const int& i, const int& j, const int& k) {
		return i > 0 && j > 0 && k > 0 && i < bounds[1].x - 1 && j < bounds[1].y - 1 && k < bounds[1].z - 1;
	}
	__host__
	friend std::ostream& operator << (std::ostream &s, const Grid &g)
	{

		s << "[ " << g.bounds[0] << std::endl
			<< g.bounds[1] << "]" << std::endl;

		return s;
	}
};

/* RAY CLASS
@Parameter
	Vector3 origin    : the ray origin point expressed as Vector3
	Vector3 direction : the ray 'end' point expressed as Vector3
@Attributes
	Vector3 orig   : as origin parameter
	Vector3 dir    : normalized vector of direction parameter
	Vector3 invDir : inverse of direction
*/
class Ray {
public:
	Vector3 orig, end, dir, length, invDir;
	__device__
	Ray(const Vector3 &startPoint, const Vector3 &endPoint) :orig(startPoint), end(endPoint) {
		length = end - orig;
		dir = length.normalize();
		invDir = 1 / dir;
	}
	__device__
	friend std::ostream& operator << (std::ostream &s, const Ray &v)
	{

		s << "[ " << std::endl
			<< "origin: " << v.orig << std::endl
			<< "end: " << v.end << std::endl
			<< "dir: " << v.dir << std::endl
			<< "invDir: " << v.invDir << std::endl
			<< "]";

		return s;
	}
};

class Camera {
public:
	Vector3 center;
	Vector3 from;
	Vector3 to;
	Vector3 forward;
	Vector3 backward;
	Vector3 right;
	Vector3 up;
	Matrix4x4 lookAt;
	Matrix4x4 worldTransformation;
	Matrix4x4 rotationM;
	double theta, phi, sigma;

	Camera() {}

	Camera(Vector3 center) {
		this->center = center;
		worldTransformation = Matrix4x4(center);

	}

	Camera(Vector3 from, Vector3 to) {
		this->from = from;
		this->to = to;
		this->rotationM = Matrix4x4();
		/*setupLookAtMatrix();*/
	}

	void yaw(const double& angle) { // z-axis
		theta = degToRad(std::fmod(angle, 360));
		Matrix4x4 rZ = Matrix4x4(std::cos(theta), -1 * std::sin(theta), 0, 0,
			std::sin(theta), std::cos(theta), 0, 0,
			0, 0, 1, 0,
			0, 0, 0, 1);
		rotationM = rotationM.matrixMulti(rZ);
	}

	void pitch(const double& angle) { //y-axis
		phi = degToRad(std::fmod(angle, 360));
		Matrix4x4 rY = Matrix4x4(std::cos(phi), 0, -1 * std::sin(phi), 0,
			0, 1, 0, 0,
			std::sin(phi), 0, std::cos(phi), 0,
			0, 0, 0, 1);
		rotationM = rotationM.matrixMulti(rY);
	}

	void roll(const double& angle) {
		sigma = degToRad(std::fmod(angle, 360));
		Matrix4x4 rX = Matrix4x4(1, 0, 0, 0,
			0, std::cos(theta), -1 * std::sin(theta), 0,
			0, std::sin(theta), std::cos(theta), 0,
			0, 0, 0, 1);
		rotationM = rotationM.matrixMulti(rX);
	}

	void setupLookAtMatrix() {
		backward = from - to;
		backward = rotationM.DirMatrixMulti(backward);
		from = to + backward;
		Vector3 tmp = rotationM.DirMatrixMulti(Vector3(0, 1, 0));
		forward = (to - from).normalize();
		right = tmp.cross(forward);
		up = forward.cross(right);
		lookAt = Matrix4x4(
			right.x, right.y, right.z, 0,
			up.x, up.y, up.z, 0,
			forward.x, forward.y, forward.z, 0,
			from.x, from.y, from.z, 1);
	}
};
__device__
inline
bool computeRayABBoxIntersection(const Ray& ray, double& tmin, double& tmax, const Grid& grid) {
	double  tminy, tmaxy, tminz, tmaxz;
	// "An efficient and robust ray-box intersection algorithm. Amy Williams et al.2004.

	if (ray.invDir.x >= 0) {
		tmin = (grid.bounds[0].x - ray.orig.x) * ray.invDir.x;
		tmax = (grid.bounds[1].x - ray.orig.x) * ray.invDir.x;
	}
	else {
		tmax = (grid.bounds[0].x - ray.orig.x) * ray.invDir.x;
		tmin = (grid.bounds[1].x - ray.orig.x) * ray.invDir.x;
	}

	if (ray.invDir.y >= 0) {
		tminy = (grid.bounds[0].y - ray.orig.y) * ray.invDir.y;
		tmaxy = (grid.bounds[1].y - ray.orig.y) * ray.invDir.y;
	}
	else {
		tmaxy = (grid.bounds[0].y - ray.orig.y) * ray.invDir.y;
		tminy = (grid.bounds[1].y - ray.orig.y) * ray.invDir.y;
	}

	if (tmin > tmaxy || tminy > tmax) return false;

	if (tminy > tmin) tmin = tminy;
	if (tmaxy < tmax) tmax = tmaxy;

	if (ray.invDir.z >= 0) {
		tminz = (grid.bounds[0].z - ray.orig.z) * ray.invDir.z;
		tmaxz = (grid.bounds[1].z - ray.orig.z) * ray.invDir.z;
	}
	else {
		tmaxz = (grid.bounds[0].z - ray.orig.z) * ray.invDir.z;
		tminz = (grid.bounds[1].z - ray.orig.z) * ray.invDir.z;
	}

	if (tmin > tmaxz || tminz > tmax) return false;

	if (tminz > tmin) tmin = tminz;
	if (tmaxz < tmax) tmax = tmaxz;

	return true;
}
__device__
double interpolation(double x, Options option) {
	if (x < option.threshold) return option.minIntensity;
	return  (x - option.minIntensity) * ((255) / (option.maxIntensity - option.minIntensity));
}
__device__
double interpolation01(double x, Options option) {
	return  (x - option.minIntensity) * ((1) / (option.maxIntensity - option.minIntensity));
}
__device__
Vector3 rasterToScreen(size_t w, size_t h, Options options) {
	// from raster to normalized to normalized to screen to camera
	double cameraX = (2 * (w + 0.5) / options.imageWidth - 1) * options.scale * options.imageAspectRatio;
	double cameraY = (1 - 2 * (h + 0.5) / options.imageHeight) * options.scale;

	return Vector3(cameraX, cameraY, -1);
}
__device__
Vector3 rasterToScreen(size_t w, size_t h, double z, Options options) {
	// from raster to normalized to normalized to screen to camera
	double cameraX = (2 * (w + 0.5) / options.imageWidth - 1) * options.imageAspectRatio;//*options.scale;
	double cameraY = (1 - 2 * (h + 0.5) / options.imageHeight);//*options.scale;

	return Vector3(cameraX, cameraY, z);
}
__device__
inline
Vector3 getGradient(const int& ix, const int& iy, const int& iz, const mxDouble * volume, const mwSize* size) {
	// component-wise linear interpolation
	double a, b, c;
	int xi = (int)(ix + 0.5);
	double xT = ix + 0.5 - xi;
	int linearXi = xi + iy * size[0] + iz * (size[0] * size[1]);
	int linearXiB = xi+1 + iy * size[0] + iz * (size[0] * size[1]);
	int linearXib = xi-1 + iy * size[0] + iz * (size[0] * size[1]);

	a = (volume[linearXi] - volume[linearXib]) * (1.0 - xT) + (volume[linearXiB] - volume[linearXi]) * xT;
	
	int yi = (int)(iy + 0.5);
	double yT = iy + 0.5 - yi;
	int linearYi = ix + yi * size[0] + iz * (size[0] * size[1]);
	int linearYiB = ix + (yi+1) * size[0] + iz * (size[0] * size[1]);
	int linearYib = ix + (yi-1) * size[0] + iz * (size[0] * size[1]);
	
	b = (volume[linearYi] - volume[linearYib]) * (1.0 - yT) + (volume[linearYiB] - volume[linearYi]) * yT;
	
	int zi = (int)(iz + 0.5);
	double zT = iz + 0.5 - zi;
	int linearZi = ix + iy * size[0] + zi * (size[0] * size[1]);
	int linearZiB = ix + iy * size[0] + (zi+1) * (size[0] * size[1]);
	int linearZib = ix + iy * size[0] + (zi-1) * (size[0] * size[1]);
	
	c = (volume[linearZi] - volume[linearZib]) * (1.0 - zT) + (volume[linearZiB] - volume[linearZi]) * zT;
	
	return Vector3(a, b, c);
}

__global__
void raycasting(mxDouble * viewOutput, mxDouble const * const volume, mxDouble const * const objectVolume,
				mwSize const * const size, int const * const visibleObj,double const *const visibleAlpha,
				Options options, Matrix4x4 lookAt, Vector3 from, 
				Vector3 lightPosition, Vector3 ambientColor,Vector3 specularColor, double shininess, double specularity,
				Color colors, YuvColor yuvColors, Matrix4x4 rgbToYuv, Matrix4x4 yuvToRgb,
				Grid grid, size_t maxStep) {
	Vector3 diffuseColor(0.5, 0.5, 0.5);
	double littleStep = 1;
	double tmin, tmax;
	int const i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < options.imageDimension) {
		int x = (i % options.imageWidth)-options.imageWidth/2;
		int y = (i / options.imageWidth)-options.imageHeight/2;
		for (size_t iObj = 0; iObj < options.visibleObjectsSize; ++iObj) {

			Vector3 rayStart = lookAt.VecMatrixMulti(Vector3(x, y, 0));
			Vector3 rayEnd = lookAt.VecMatrixMulti(Vector3(x, y, 1));

			Ray ray(rayStart, rayEnd);

			if (computeRayABBoxIntersection(ray, tmin, tmax, grid)) {

				Vector3 start = ray.orig + ray.dir *tmin;
				Vector3 end = ray.orig + ray.dir *tmax;

				for (size_t t = 0; t < maxStep; t += littleStep) {
					int ix = floor(start.x) > 0 ? floor(start.x) - 1 : 0;
					int iy = floor(start.y) > 0 ? floor(start.y) - 1 : 0;
					int iz = floor(start.z) > 0 ? floor(start.z) - 1 : 0;
					int linearIndex = ix + iy * size[0] + iz * (size[0] * size[1]);
					if (grid.isInsideGrid(ix, iy, iz)) {
						if (volume[linearIndex] >= options.threshold && (visibleObj[iObj] == -1|| objectVolume[linearIndex] == visibleObj[iObj])) {

							Vector3 grad = getGradient(ix, iy, iz, volume,size);
							Vector3 normal = -grad / std::sqrt(grad.norm());

							Vector3 lightDir = from.normalize();
							double distance = lightPosition.length();
							double lambertian = lightDir.dot(normal);

							Vector3 viewDir = rayStart.normalize();
							Vector3 halfDir = (lightDir + viewDir).normalize();

							double specAngle = halfDir.dot(normal);
							double specular = std::pow(specAngle, shininess);

							diffuseColor = colors.getColorByIndex(visibleObj[iObj]);

							// 1* lightAmbientColor  +  1* lightDiffuseColor*dot(lightdir,normals) * weight  +  1* lightSpecularColor * [dot(halfdir,normals)]^shininess * (1-weight)
							Vector3 IlluminationI = ambientColor + diffuseColor * lambertian * specularity + specularColor * specular * (1 - specularity);
							//convert to yuv
							Vector3 yuvIllumination = rgbToYuv.ColMajMatrixMulti(IlluminationI);
							//get diffuse color in yuv color schema
							Vector3 yuvDiffuse = yuvColors.getColorByIndex(visibleObj[iObj]);
							//reset color
							yuvIllumination.y = yuvDiffuse.y;
							yuvIllumination.z = yuvDiffuse.z;
							//convert back to rgb
							IlluminationI = yuvToRgb.ColMajMatrixMulti(yuvIllumination);

							// clamp new illumination to valid rgb value.ie: x<0-> x=0, x>1 -> x=1
							viewOutput[i] += visibleAlpha[iObj] * IlluminationI.x;
							viewOutput[i + options.imageDimension] += visibleAlpha[iObj] * IlluminationI.y;
							viewOutput[i + options.imageDimension * 2] += visibleAlpha[iObj] * IlluminationI.z;

							/*viewOutput[i] = 0.7;
							viewOutput[i + options.imageDimension] = 0.5;
							viewOutput[i + options.imageDimension * 2] = 0.3;*/
							break;
						}
					}
					start = start + ray.dir*littleStep;

				}

			}
		}
	}
	return;
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	std::time(&timer);
	Options options;
	/*
	* GPU INIT
	* CREATING OR REFERINCING VARIABLE IN DEVICE MEMORY
	*/
	mxInitGPU();

	mxGPUArray const * d_VolumeArray = mxGPUCreateFromMxArray(prhs[0]);
	const mxDouble *  d_Volume = (mxDouble const*)mxGPUGetDataReadOnly(d_VolumeArray);
	const mwSize *sizeArray = mxGetDimensions(prhs[0]);
	const mwSize numberOfVoxels = mxGetNumberOfElements(prhs[0]);
	mwSize * d_Size;
	hipMalloc((void**)&d_Size, 3 * sizeof(mwSize));
	hipMemcpy(d_Size, sizeArray, 3 * sizeof(mwSize), hipMemcpyHostToDevice);

	mxGPUArray const * d_ObjectsArray = mxGPUCreateFromMxArray(prhs[1]);
	const mxDouble *  d_ObjectVolume = (mxDouble const*)mxGPUGetDataReadOnly(d_ObjectsArray);
	const mwSize *objectSizeArray = mxGetDimensions(prhs[0]);
	const mwSize numberOfObjects = mxGetNumberOfElements(prhs[1]);

	const mxDouble *thresholdArray = mxGetPr(mxGetField(prhs[2], 0, "threshold"));
	const mxDouble *viewArray = mxGetPr(mxGetField(prhs[2], 0, "view"));
	const mxDouble *alphaArray = mxGetPr(mxGetField(prhs[2], 0, "alpha"));
	const mxDouble *specularityArray = mxGetPr(mxGetField(prhs[2], 0, "specularity"));
	const mxDouble *intensityArray = mxGetPr(mxGetField(prhs[2], 0, "intensity"));
	const mxDouble *rotationArray = mxGetPr(mxGetField(prhs[2], 0, "rotation"));
	const mxDouble *colorArray = mxGetPr(mxGetField(prhs[2], 0, "colors"));

	const mwSize alphaSize = mxGetNumberOfElements(mxGetField(prhs[2], 0, "alpha"));

	options.setImageSize(viewArray[0], viewArray[1]);
	options.imageDimension = options.imageHeight * options.imageWidth;
	options.setIntensity(intensityArray[0], intensityArray[1]);
	options.threshold = (thresholdArray[0]);
	options.fov = 52.51;
	options.viewOffset = 200;
	options.scale = std::tan(degToRad((options.fov * 0.5)));
	options.imageAspectRatio = options.imageWidth / options.imageHeight;

	mwSize frameDimensions[3] = { options.imageWidth,options.imageHeight,3 };

	mxGPUArray * const  d_viewArray = mxGPUCreateGPUArray(3, frameDimensions, mxDOUBLE_CLASS, mxREAL, MX_GPU_INITIALIZE_VALUES);
	mxDouble * d_viewOutput = (mxDouble *)mxGPUGetData(d_viewArray);
#if DEBUG
	std::cout << "SIZE " << sizeArray[0] << " " << sizeArray[1] << " " << sizeArray[2] << std::endl
		<< "VOX " << numberOfVoxels << std::endl
		<< "VIEW " << viewArray[0] << " " << viewArray[1] << std::endl
		<< "ROT " << rotationArray[0] << " " << rotationArray[1] << " " << rotationArray[2] << std::endl
		<< "INT " << intensityArray[0] << " " << intensityArray[1] << std::endl
		<< "THR " << thresholdArray[0] << std::endl
		<< "OBJS SIZE " << objectSizeArray[0] << " " << objectSizeArray[1] << " " << objectSizeArray[2] << std::endl;
#endif

	if (numberOfObjects > 0) {
		for (int j = 0; j < alphaSize; ++j) {
			if (alphaArray[j] > 0.0) {
				options.visibleObjectsSize++;
			}
		}
	}
	else {
		options.visibleObjectsSize = 1;
	}
	
	int * visibleObj = new int[options.visibleObjectsSize];
	double * visibleAlpha = new double[options.visibleObjectsSize];

	if (numberOfObjects == 0) {
		visibleAlpha[0] = 1;
		visibleObj[0] = -1;
	}
	else {
		int visObj = 0;
		for (size_t i = 0; i < alphaSize; ++i) {
			if (alphaArray[i] > 0) {
				visibleAlpha[visObj] = alphaArray[i];
				visibleObj[visObj] = i;
				visObj++;
			}
		}
	}

	Matrix4x4 rgbToYuv(0.2126, 0.7152, 0.0722, 0, -0.09991, -0.33609, 0.436, 0, 0.615, -0.55861, -0.05639, 0, 0, 0, 0, 0);
	Matrix4x4 yuvToRgb(1, 0, 1.28033, 0, 1, -0.21482, -0.38059, 0, 1, 2.12798, 0, 0, 0, 0, 0, 0);

	Vector3 lightPosition(sizeArray[0] / 2 + options.viewOffset, sizeArray[1] / 2, sizeArray[2] / 2);

	Vector3 ambientColor(0.5, 0.5, 0.5);
	Vector3 diffuseColor(0.6, 0.6, 0.6);
	Vector3 specularColor(0.7, 0.7, 0.7);

	double shininess = 16.0;
	double specularity = specularityArray[0];

	Grid grid(Vector3(0), Vector3(sizeArray[0], sizeArray[1], sizeArray[2]));

	Camera camera(Vector3(sizeArray[0] / 2 + options.viewOffset, sizeArray[1] / 2, sizeArray[2] / 2), Vector3(sizeArray[0] / 2, sizeArray[1] / 2, sizeArray[2] / 2));

	camera.pitch(rotationArray[1]);
	camera.roll(rotationArray[2]);
	camera.yaw(rotationArray[0]);
	camera.setupLookAtMatrix();

	size_t maxStep = std::sqrt(sizeArray[0] * sizeArray[0] + sizeArray[1] * sizeArray[1] + sizeArray[2] * sizeArray[2]);

	double halfWidth = std::fabs(options.imageWidth / 2);
	double halfHeight = std::fabs(options.imageHeight / 2);

	Color colors;
	YuvColor yuvColors;

	hipError_t err = hipSuccess;
	int* d_visibleObj;
	hipMalloc((void**)&d_visibleObj, options.visibleObjectsSize * sizeof(int));
	err = hipMemcpy(d_visibleObj, visibleObj, options.visibleObjectsSize * sizeof(int), hipMemcpyHostToDevice);
	double* d_visibleAlpha;
	hipMalloc((void**)&d_visibleAlpha, options.visibleObjectsSize * sizeof(double));
	err = hipMemcpy(d_visibleAlpha, visibleAlpha, options.visibleObjectsSize * sizeof(double), hipMemcpyHostToDevice);

	//call kernel
	raycasting<<<std::ceil(options.imageDimension / 256.0), 256 >>>(d_viewOutput, d_Volume, d_ObjectVolume, d_Size, d_visibleObj,d_visibleAlpha,
																	options, camera.lookAt, camera.from,
																	lightPosition,ambientColor,specularColor,shininess,specularity,
																	colors,yuvColors,rgbToYuv,yuvToRgb,
																	grid, maxStep);

	// uncomment to show GPU work
	plhs[0] = mxGPUCreateMxArrayOnGPU(d_viewArray);
	delete [] visibleObj;
	delete [] visibleAlpha;
	hipFree(d_visibleAlpha);
	hipFree(d_visibleObj);
	hipFree(d_Size);
	mxGPUDestroyGPUArray(d_viewArray);
	mxGPUDestroyGPUArray(d_ObjectsArray);
	mxGPUDestroyGPUArray(d_VolumeArray);
	return;
}